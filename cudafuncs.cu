#include "hip/hip_runtime.h"
#include "cudafuncs.hpp"

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err!=hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

__global__ void bgr_to_gray_kernel( unsigned char* input,
									unsigned char* output,
									int width,
									int height,
									int colorWidthStep,
									int grayWidthStep)
{
	//2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if((xIndex<width) && (yIndex<height))
	{
		//Location of colored pixel in input
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		
		//Location of gray pixel in output
		const int gray_tid  = yIndex * grayWidthStep + xIndex;

		const unsigned char blue	= input[color_tid];
		const unsigned char green	= input[color_tid + 1];
		const unsigned char red		= input[color_tid + 2];

		const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;

		output[gray_tid] = static_cast<unsigned char>(gray);
	}
}

void convert_to_gray(const cv::Mat& input, cv::Mat& output)
{
	//Calculate total number of bytes of input and output image
	const int colorBytes = input.step * input.rows;
	const int grayBytes = output.step * output.rows;

	unsigned char *d_input, *d_output;

	//Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input,colorBytes),"CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output,grayBytes),"CUDA Malloc Failed");

	//Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input,input.ptr(),colorBytes,hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");

	//Specify a reasonable block size
	const dim3 block(16,16);

	//Calculate grid size to cover the whole image
	const dim3 grid((input.cols + block.x - 1)/block.x, (input.rows + block.y - 1)/block.y);

	//Launch the color conversion kernel
	bgr_to_gray_kernel<<<grid,block>>>(d_input,d_output,input.cols,input.rows,input.step,output.step);

	//Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(),"Kernel Launch Failed");

	//Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(),d_output,grayBytes,hipMemcpyDeviceToHost),"CUDA Memcpy Host To Device Failed");

	//Free the device memory
	SAFE_CALL(hipFree(d_input),"CUDA Free Failed");
	SAFE_CALL(hipFree(d_output),"CUDA Free Failed");
}
